#include "hip/hip_runtime.h"
﻿#include "image_handler.h"
#include "seam_carving.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

__global__
void fillMinSeam_(seam_t* minSeam, int width) {
	int idThread = blockIdx.x * blockDim.x + threadIdx.x;

	minSeam->ids[idThread] = idThread * width;

}

void applySeamCarving(char *p) {

	pixel_t* imgSrc;
	imgProp_t* imgProp;
	energyPixel_t* imgGray;
	energyPixel_t* imgEnergy;
	seam_t* minSeam;

	FILE* f = fopen(p, "rb");
	if (f == NULL) {
		printf("*** FILE NOT FOUND ***\n");
		exit(1);
	}

	gpuErrchk(hipMallocManaged(&imgProp, sizeof(imgProp_t)));
	setupImgProp(imgProp, f);

	gpuErrchk(hipMallocManaged(&imgSrc, imgProp->height * imgProp->width * sizeof(pixel_t)));
	gpuErrchk(hipMallocManaged(&imgGray, imgProp->height * imgProp->width * sizeof(energyPixel_t)));
	gpuErrchk(hipMallocManaged(&imgEnergy, imgProp->height * imgProp->width * sizeof(energyPixel_t)));

	gpuErrchk(hipMallocManaged(&minSeam, sizeof(seam_t)));
	gpuErrchk(hipMallocManaged(&minSeam->ids, imgProp->height * sizeof(int)));
	int numBlocks = imgProp->width / 1024 + 1;
	seam_t* seams;
	seam_t* minSeamsPerBlock;

	gpuErrchk(hipMallocManaged(&seams, imgProp->width * sizeof(seam_t)));
	for (int i = 0; i < imgProp->width; i++)
		gpuErrchk(hipMallocManaged(&seams[i].ids, imgProp->height * sizeof(int)));

	gpuErrchk(hipMallocManaged(&minSeamsPerBlock, numBlocks * sizeof(seam_t)));
	for (int i = 0; i < numBlocks; i++)
		gpuErrchk(hipMallocManaged(&minSeamsPerBlock[i].ids, imgProp->height * sizeof(int)));
	

	readBMP(f, imgSrc, imgProp);
	//writeBMP_pixel(strcat(SOURCE_PATH, "hhh.bmp"), imgSrc, imgProp);
	toGrayScale(imgSrc, imgGray, imgProp);
	
	for (int i = 0; i < 100; i++) {
		map(imgGray, imgProp);
		//printf("-----------------width %d height %d\n", imgProp->width, imgProp->height);
		
		findSeams(imgGray, imgProp, minSeam, seams, minSeamsPerBlock);
		
		//fillMinSeam_ << <1, 968>> > (minSeam, imgProp->width);
		//hipDeviceSynchronize();
		
		//for (int i = 0; i < imgProp->height; i++) {
		//	printf("%d - ", minSeam[0].ids[i]);
		//}
		removeSeam(imgGray, minSeam, imgProp);
		printf("ITERAZIONE %d COMPLETATA\n", i);
	}
	setBMP_header(imgProp, 0, imgProp->width);

	pixel_t* img2convert = (pixel_t*)malloc(imgProp->imageSize * sizeof(pixel_t));
	energy2pixel(img2convert, imgGray, imgProp);
	writeBMP_pixel(strcat(SOURCE_PATH, "ffff.bmp"), img2convert, imgProp);
	
	free(img2convert);
	

	hipFree(imgProp);
	hipFree(imgGray);
	hipFree(imgSrc);
	//hipFree(seams);
	//hipFree(minSeamsPerBlock);
	fclose(f);
}



int main(int argc, char** argv) {

	/*
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
	*/

	//imgProp_t* imgProp;

	char* path = strcat(SOURCE_PATH, "castle_bmp.bmp");

	applySeamCarving(path);

	//hipMallocManaged(imgGray, imgProp->imageSize);
	//map(imgGray, imgProp);
	hipDeviceReset();
	return 0;
}
