﻿#include "image_handler.h"
#include "seam_carving.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

void applySeamCarving(char *p) {

	pixel_t* imgSrc;
	imgProp_t* imgProp;
	energyPixel_t* imgGray;
	energyPixel_t* imgEnergy;
	seam_t* minSeam;

	FILE* f = fopen(p, "rb");
	if (f == NULL) {
		printf("*** FILE NOT FOUND ***\n");
		exit(1);
	}

	hipMallocManaged(&imgProp, sizeof(imgProp_t));
	setupImgProp(imgProp, f);

	hipMallocManaged(&imgSrc, imgProp->height * imgProp->width * sizeof(pixel_t));
	hipMallocManaged(&imgGray, imgProp->height * imgProp->width * sizeof(energyPixel_t));
	hipMallocManaged(&imgEnergy, imgProp->height * imgProp->width * sizeof(energyPixel_t));

	hipMallocManaged(&minSeam, sizeof(seam_t));
	hipMallocManaged(&minSeam->ids, imgProp->height * sizeof(int));

	readBMP(f, imgSrc, imgProp);
	//writeBMP_pixel(strcat(SOURCE_PATH, "hhh.bmp"), imgSrc, imgProp);
	toGrayScale(imgSrc, imgGray, imgProp);
	for (int i = 0; i < 1; i++) {
		map(imgGray, imgProp);
		printf("-----------------width %d height %d\n", imgProp->width, imgProp->height);
		findSeams(imgGray, imgProp, minSeam);
		removeSeam(imgGray, minSeam->ids, imgProp);
		printf("ITERAZIONE %d COMPLETATA\n", i);
	}

	setBMP_header(imgProp, 0, imgProp->width);

	pixel_t* img2convert = (pixel_t*)malloc(imgProp->imageSize * sizeof(pixel_t));
	energy2pixel(img2convert, imgGray, imgProp);
	writeBMP_pixel(strcat(SOURCE_PATH, "ffff.bmp"), img2convert, imgProp);
	free(img2convert);

	hipFree(imgProp);
	hipFree(imgGray);
	hipFree(imgSrc);

	fclose(f);
}

int main(int argc, char** argv) {

	/*
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
	*/

	//imgProp_t* imgProp;

	char* path = strcat(SOURCE_PATH, "33.bmp");

	applySeamCarving(path);

	//hipMallocManaged(imgGray, imgProp->imageSize);
	//map(imgGray, imgProp);
	hipDeviceReset();
	return 0;
}

//void report_gpu_mem()
//{
//	size_t free, total;
//	hipMemGetInfo(&free, &total);
//	printf("Free = %zu, Total = %zu\n", free, total);
//}
//
//int main()
//{
//	float* a, * a_out, *b, *bo;
//	int sz = 1 << 20; // 16Mb
//	report_gpu_mem();
//	hipMallocManaged((void**)&a, sz);
//	report_gpu_mem();
//	hipMallocManaged((void**)&a_out, sz);
//	report_gpu_mem();
//
//	hipMallocManaged((void**)&b, sz);
//	report_gpu_mem();
//	hipMallocManaged((void**)&bo, sz);
//	report_gpu_mem();
//
//
//	hipFree(a);
//	report_gpu_mem();
//	hipFree(a_out);
//	report_gpu_mem();
//	hipFree(b);
//	report_gpu_mem();
//	hipFree(bo);
//	report_gpu_mem();
//	return hipDeviceReset();
//}