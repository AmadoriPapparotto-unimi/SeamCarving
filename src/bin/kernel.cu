﻿#include "image_handler.h"
#include "seam_carving.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

char* src_path;

void applySeamCarving(char *p, int iterations) {

	imgProp_t* imgProp;

	pixel_t* imgSrc;
	pixel_t* imgWithoutSeamSrc;
	
	energyPixel_t* imgGray;
	energyPixel_t* imgWithoutSeamGray;
	
	seam_t* seams;
	seam_t* minSeamsPerBlock;
	seam_t* minSeam;

	FILE* f = fopen(p, "rb");
	if (f == NULL) {
		printf("*** FILE NOT FOUND ***\n");
		exit(1);
	}

	gpuErrchk(hipMallocManaged(&imgProp, sizeof(imgProp_t)));
	setupImgProp(imgProp, f);

	int numBlocks = imgProp->width / 1024 + 1;

	gpuErrchk(hipMallocManaged(&imgSrc, imgProp->imageSize * sizeof(pixel_t)));
	gpuErrchk(hipMallocManaged(&imgGray, imgProp->imageSize * sizeof(energyPixel_t)));
	gpuErrchk(hipMallocManaged(&imgWithoutSeamSrc, imgProp->imageSize * sizeof(pixel_t)));
	gpuErrchk(hipMallocManaged(&imgWithoutSeamGray, imgProp->imageSize * sizeof(energyPixel_t)));

	gpuErrchk(hipMallocManaged(&seams, imgProp->width * sizeof(seam_t)));
	for (int i = 0; i < imgProp->width; i++)
		gpuErrchk(hipMallocManaged(&seams[i].ids, imgProp->height * sizeof(int)));

	gpuErrchk(hipMallocManaged(&minSeamsPerBlock, numBlocks * sizeof(seam_t)));
	for (int i = 0; i < numBlocks; i++)
		gpuErrchk(hipMallocManaged(&minSeamsPerBlock[i].ids, imgProp->height * sizeof(int)));

	gpuErrchk(hipMallocManaged(&minSeam, sizeof(seam_t)));
	gpuErrchk(hipMallocManaged(&minSeam->ids, imgProp->height * sizeof(int)));

	readBMP(f, imgSrc, imgProp);
	toGrayScale(imgSrc, imgGray, imgProp);
	
	for (int i = 0; i < iterations; i++) {
		energyMap(imgGray, imgProp);		
		findSeams(imgGray, imgSrc, imgProp, minSeam, seams, minSeamsPerBlock);
		removeSeam(imgGray, imgSrc, imgWithoutSeamGray, imgWithoutSeamSrc, minSeam, imgProp);
		printf("ITERAZIONE %d COMPLETATA\n", i);
	}

	setBMP_header(imgProp, 0, imgProp->width);
	writeBMP_pixel(strcat(SOURCE_PATH, "reduced.bmp"), imgSrc, imgProp);
		
	fclose(f);
}



int main(int argc, char** argv) {

	/*
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
	*/

	char* path = argv[1];
	int iterations = atoi(argv[2]);

	applySeamCarving(path, iterations);
	hipDeviceReset();

	return 0;
}
