﻿#include "image_handler.h"
#include "seam_carving.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

void applySeamCarving(char *p) {

	pixel_t* imgSrc;
	imgProp_t* imgProp;
	energyPixel_t* imgGray;

	FILE* f = fopen(p, "rb");
	if (f == NULL) {
		printf("*** FILE NOT FOUND ***\n");
		exit(1);
	}

	hipMallocManaged(&imgProp, sizeof(imgProp_t));
	setupImgProp(imgProp, f);

	hipMallocManaged(&imgSrc, imgProp->height * imgProp->width * sizeof(pixel_t));
	hipMallocManaged(&imgGray, imgProp->height * imgProp->width * sizeof(energyPixel_t));

	readBMP(f, imgSrc, imgProp);
	//writeBMP_pixel(strcat(SOURCE_PATH, "hhh.bmp"), imgSrc, imgProp);
	toGrayScale(imgSrc, imgGray, imgProp);
	map(imgGray, imgProp);
	findSeams(imgGray, imgProp);

	hipFree(imgProp);
	hipFree(imgGray);
	hipFree(imgSrc);

	fclose(f);
}

int main(int argc, char** argv) {

	/*
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
	*/

	//imgProp_t* imgProp;

	char* path = strcat(SOURCE_PATH, "a.bmp");

	applySeamCarving(path);

	//hipMallocManaged(imgGray, imgProp->imageSize);
	//map(imgGray, imgProp);

	return 0;
}