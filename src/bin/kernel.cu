﻿#include "image_handler.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

int main(int argc, char** argv) {
	pixel* imgSrc;				

	//imgSrc = ReadBMP(argv[1]);
	imgSrc = readBMP("src/assets/images/castle_bmp.bmp");

	if (imgSrc == NULL) {
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	return 0;
}