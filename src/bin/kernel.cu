﻿#include "image_handler.h"
#include "seam_carving.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

pixel_t* imgSrc;
energyPixel_t* imgGray;

int main(int argc, char** argv) {

	imgProp_t* imgProp;
	hipMallocManaged(&imgProp, sizeof(imgProp_t));

	char* path = strcat(SOURCE_PATH, "castle_bmp.bmp");

	readBMP(imgSrc, imgGray, path, imgProp);
	//hipMallocManaged(imgGray, imgProp->imageSize);
	//map(imgGray, imgProp);

	return 0;
}