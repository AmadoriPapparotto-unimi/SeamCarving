﻿#include "image_handler.h"
#include "seam_carving.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

char* src_path;

void applySeamCarving(char *p, int iterations) {

	pixel_t* imgSrc;
	imgProp_t* imgProp;
	energyPixel_t* imgGray;
	energyPixel_t* imgEnergy;
	energyPixel_t* imgWithoutSeam;
	seam_t* minSeam;

	FILE* f = fopen(p, "rb");
	if (f == NULL) {
		printf("*** FILE NOT FOUND ***\n");
		exit(1);
	}

	gpuErrchk(hipMallocManaged(&imgProp, sizeof(imgProp_t)));
	setupImgProp(imgProp, f);

	gpuErrchk(hipMallocManaged(&imgSrc, imgProp->height * imgProp->width * sizeof(pixel_t)));
	gpuErrchk(hipMallocManaged(&imgGray, imgProp->height * imgProp->width * sizeof(energyPixel_t)));
	gpuErrchk(hipMallocManaged(&imgEnergy, imgProp->height * imgProp->width * sizeof(energyPixel_t)));

	gpuErrchk(hipMallocManaged(&minSeam, sizeof(seam_t)));
	gpuErrchk(hipMallocManaged(&minSeam->ids, imgProp->height * sizeof(int)));
	int numBlocks = imgProp->width / 1024 + 1;
	seam_t* seams;
	seam_t* minSeamsPerBlock;

	gpuErrchk(hipMallocManaged(&seams, imgProp->width * sizeof(seam_t)));
	for (int i = 0; i < imgProp->width; i++)
		gpuErrchk(hipMallocManaged(&seams[i].ids, imgProp->height * sizeof(int)));

	gpuErrchk(hipMallocManaged(&minSeamsPerBlock, numBlocks * sizeof(seam_t)));
	for (int i = 0; i < numBlocks; i++)
		gpuErrchk(hipMallocManaged(&minSeamsPerBlock[i].ids, imgProp->height * sizeof(int)));
		gpuErrchk(hipMallocManaged(&imgWithoutSeam, imgProp->imageSize * sizeof(energyPixel_t)));


	readBMP(f, imgSrc, imgProp);
	//writeBMP_pixel(strcat(SOURCE_PATH, "hhh.bmp"), imgSrc, imgProp);
	toGrayScale(imgSrc, imgGray, imgProp);
	
	for (int i = 0; i < iterations; i++) {
		energyMap(imgGray, imgProp);		
		findSeams(imgGray, imgProp, minSeam, seams, minSeamsPerBlock);
		removeSeam(imgGray, imgWithoutSeam, minSeam, imgProp);
		printf("ITERAZIONE %d COMPLETATA\n", i);
	}

	setBMP_header(imgProp, 0, imgProp->width);

	pixel_t* img2convert = (pixel_t*)malloc(imgProp->imageSize * sizeof(pixel_t));
	energy2pixel(img2convert, imgGray, imgProp);
	writeBMP_pixel(strcat(SOURCE_PATH, "ffff.bmp"), img2convert, imgProp);
	
	free(img2convert);
	

	//hipFree(imgProp);
	//hipFree(imgGray);
	//hipFree(imgSrc);
	//gpuErrchk(hipFree(imgWithoutSeam));
	// 
	//hipFree(seams);
	//hipFree(minSeamsPerBlock);
	fclose(f);
}



int main(int argc, char** argv) {

	/*
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
	*/

	char* path = argv[1];//strcat(SOURCE_PATH, "big.bmp");
	int iterations = atoi(argv[2]);
	//src_path = path
	applySeamCarving(path, iterations);
	hipDeviceReset();

	return 0;
}
