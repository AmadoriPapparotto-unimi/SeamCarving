#include "hip/hip_runtime.h"
﻿#include "image_handler.h"
#include "seam_carving.h"
#include "utils.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#define MAX_THREAD 1024


__device__ void grayValue(pixel_t *res, pel_t r, pel_t g, pel_t b) {
	int grayVal = (r + g + b) / 3;
	res->R = grayVal;
	res->G = grayVal;
	res->B = grayVal;
}

__global__ void toGrayScale(pixel_t* img, energyPixel_t* imgGray, int imageSize)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == gridDim.x * 1024 + 1)
		printf("%d", gridDim.x);

	if (id < imageSize) {
		grayValue(&imgGray[id].pixel, img[id].R, img[id].G, img[id].B);
	}
}

void setupImgProp(imgProp_t* ip, FILE* f) {
	pel_t headerInfo[54];
	fread(headerInfo, sizeof(pel_t), 54, f);

	int width = *(int*)&headerInfo[18];
	int height = *(int*)&headerInfo[22];
	printf("#bytes: %d\n", *(int*)&headerInfo[34]);

	for (unsigned int i = 0; i < 54; i++)
		ip->headerInfo[i] = headerInfo[i];

	ip->height = height;
	ip->width = width;
	ip->imageSize = width * height;
}

void readBMP(pixel_t* img, energyPixel_t* imgGray, char* p, imgProp_t* ip) {

	//img[0] = B
	//img[1] = G
	//img[2] = R
	//BMP LEGGE I PIXEL NEL FORMATO BGR
	FILE* f = fopen(p, "rb");
	if (f == NULL) {
		printf("*** FILE NOT FOUND ***\n");
		exit(1);
	}

	//extract information from headerInfo
	setupImgProp(ip, f);
	printf("Input BMP dimension: (%u x %u)\n", ip->width, ip->height);

	hipMallocManaged(&img, ip->height * ip->width * sizeof(pixel_t));
	hipMallocManaged(&imgGray, ip->height * ip->width * sizeof(energyPixel_t));

	for (unsigned int i = 0; i < ip->height * ip->width; i++) {
		fread(&img[i], sizeof(pel_t), sizeof(pixel_t), f);
	}

	dim3 blocks;
	blocks.x = ip->imageSize / MAX_THREAD + 1;

	toGrayScale << <blocks, MAX_THREAD >> > (img, imgGray, ip->imageSize);
	hipDeviceSynchronize();
	writeBMP_pixel(strcat(SOURCE_PATH, "gray.bmp"), energy2pixel(imgGray, ip), ip);

	fclose(f);

	map(imgGray, ip);
	findSeams(imgGray, ip);

}

void writeBMP_pixel(char* p, pixel_t* img, imgProp_t* ip) {
	FILE* fw = fopen(p, "wb");

	fwrite(ip->headerInfo, sizeof(pel_t), 54, fw);
	fwrite(img, sizeof(pixel_t), ip->imageSize, fw);

	fclose(fw);
	printf("Immagine %s generata\n", p);
}

void writeBMP_energy(char* p, energyPixel_t* energyImg, imgProp_t* ip) {
	pixel_t* img;
	int sd = 1;
	img = (pixel_t*)malloc(ip->imageSize * sizeof(pixel_t));

	for (int i = 0; i < ip->imageSize; i++) {
		img[i].R = energyImg[i].energy;
		img[i].G = energyImg[i].energy;
		img[i].B = energyImg[i].energy;
	}

	writeBMP_pixel(p, img, ip);
}

void writeBMP_minimumSeam(char* p, energyPixel_t* energyImg, seam_t* minSeam, imgProp_t* imgProp) {
	for (int y = 0; y < imgProp->height; y++) {
		printf("PATH: %d\n", minSeam[0].ids[y]);
		energyImg[minSeam[0].ids[y]].pixel.R = 0;
		energyImg[minSeam[0].ids[y]].pixel.G = 255;
		energyImg[minSeam[0].ids[y]].pixel.B = 0;
	}
	writeBMP_pixel(strcat(SOURCE_PATH, "seams_map_minimum.bmp"), energy2pixel(energyImg, imgProp), imgProp);
}

pixel_t* energy2pixel(energyPixel_t* energyImg, imgProp_t* ip) {
	pixel_t* img;
	img = (pixel_t*)malloc(ip->imageSize * sizeof(pixel_t));

	for (int i = 0; i < ip->imageSize; i++) {
		img[i] = energyImg[i].pixel;
	}

	return img;
}

//void writeBMP_pel(char* p, imgProp imgProp, pel* img) {
//	FILE* fw = fopen(p, "wb");
//
//	//0000 0000 0001 0101 0001 0111 1010 0000
//	imgProp.headerInfo[34] = ip.imageSize >> 0;//0xa0;
//	imgProp.headerInfo[35] = ip.imageSize >> 8;//0x17;
//	imgProp.headerInfo[36] = ip.imageSize >> 16;//0x15;
//	imgProp.headerInfo[37] = ip.imageSize >> 24;//0x0;
//	
//	printf("%ld; %d", imgProp.headerInfo[34], imgProp.height * imgProp.width);
//	fwrite(imgProp.headerInfo, sizeof(pel), 54, fw);
//	fwrite(img, sizeof(pel), imgProp.height * imgProp.width, fw);
//
//	fclose(fw);
//}