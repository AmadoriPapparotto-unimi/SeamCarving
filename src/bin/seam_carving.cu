#include "hip/hip_runtime.h"
﻿#include "image_handler.h"
#include "seam_carving.h"
#include "math.h"
#include "utils.h"

#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

__device__ char getPosition(int id, imgProp_t* imgProp) {
	int width = imgProp->width;
	int imageSize = imgProp->imageSize;

	//ANGOLO BASSO SX:									[0]
	//ANGOLO ALTO SX									[1]
	//COLONNA SX: id % imgProp->width == 0				[2]
	//ANGOLO BASSO DX									[3]
	//ANGOLO ALTO DX									[4]
	//COLONNA DX: id + 1 %  imgProp->width == 0			[5]
	//RIGA DOWN: id < imgProp->width					[6]
	//RIGA UP: id > imgProp.imageSize - imgProp->width	[7]
	//IN MEZZO											[-1]
	/*
		08 09 10 11
		04 05 06 07
		00 01 02 03
	*/

	if (id % width == 0) {
		if (id == 0)
			return 0;
		if (id == imageSize - width)
			return 1;
		return 2;
	}
	else if (id % width == width - 1) {
		if (id == width - 1)
			return 3;
		if (id == imageSize - 1)
			return 4;
		return 5;
	}
	else if (id < width)
		return 6;
	else if (id > imageSize - width)
		return 7;
	return -1;
}

__device__ void calculateEnergy(energyPixel_t* energyPixel, energyPixel_t* pixel, int id, imgProp_t* imgProp) {
	int dx2, dy2;
	//ANGOLO BASSO SX:									[0]
	//ANGOLO ALTO SX									[1]
	//COLONNA SX: id % imgProp->width == 0				[2]
	//ANGOLO BASSO DX									[3]
	//ANGOLO ALTO DX									[4]
	//COLONNA DX: id + 1 %  imgProp->width == 0			[5]
	//RIGA DOWN: id < imgProp->width					[6]
	//RIGA UP: id > imgProp.imageSize - imgProp->width	[7]
	//IN MEZZO											[-1]
	/*
		678
		345
		012
	*/
	int inde = 1382303 - (1428) - (1428 - 540);

	char pos = getPosition(id, imgProp);
	switch (pos)
	{
	case 0:
		dx2 = energyPixel[id + 1].color;
		dy2 = energyPixel[id + imgProp->width].color;
		break;
	case 1:
		dx2 = energyPixel[id + 1].color;
		dy2 = energyPixel[id - imgProp->width].color;
		break;
	case 2:
		dx2 = energyPixel[id + 1].color;
		dy2 = energyPixel[id + imgProp->width].color - energyPixel[id - imgProp->width].color;
		break;
	case 3:
		dx2 = energyPixel[id - 1].color;
		dy2 = energyPixel[id + imgProp->width].color;
		break;
	case 4:
		dx2 = energyPixel[id - 1].color;
		dy2 = energyPixel[id - imgProp->width].color;
		break;
	case 5:
		dx2 = energyPixel[id - 1].color;
		dy2 = energyPixel[id + imgProp->width].color - energyPixel[id - imgProp->width].color;
		break;
	case 6:
		dx2 = energyPixel[id - 1].color - energyPixel[id + 1].color;
		dy2 = energyPixel[id + imgProp->width].color;
		break;
	case 7:
		dx2 = energyPixel[id - 1].color - energyPixel[id + 1].color;
		dy2 = energyPixel[id - imgProp->width].color;
		break;
	case -1:
		dx2 = energyPixel[id - 1].color - energyPixel[id + 1].color;
		dy2 = energyPixel[id + imgProp->width].color - energyPixel[id - imgProp->width].color;
		break;
	}

	pixel->energy = sqrtf((dx2 * dx2) + (dy2 * dy2));
}

__global__ void energyMap_(energyPixel_t* energyImg, imgProp_t* imgProp) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < imgProp->imageSize) {
		calculateEnergy(energyImg, &energyImg[id], id, imgProp);
	}
}

__device__ int min(int id1, int id2, energyPixel_t* energyImg)
{
	return (energyImg[id1].energy < energyImg[id2].energy) ? id1 : id2;
}

__global__ void computeSeams(energyPixel_t* energyImg, pixel_t* imgSrc, seam_t* seams, imgProp_t* imgProp, bool colorSeams = false) {

	//ANGOLO BASSO SX:									[0]
	//ANGOLO ALTO SX									[1]
	//COLONNA SX: id % imgProp->width == 0				[2]
	//ANGOLO BASSO DX									[3]
	//ANGOLO ALTO DX									[4]
	//COLONNA DX: id + 1 %  imgProp->width == 0			[5]
	//RIGA DOWN: id < imgProp->width					[6]
	//RIGA UP: id > imgProp.imageSize - imgProp->width	[7]
	//IN MEZZO											[-1]
	/*
		678
		345
		012
	*/

	int idThread = blockIdx.x * blockDim.x + threadIdx.x;

	int currentId = idThread;
	if (currentId > imgProp->width - 1)
		return;
	int nextIdMin = currentId;


	seams[idThread].total_energy = 0;
	//seams[idThread].ids[0] = currentId;



	for (int i = 0; i < imgProp->height; i++) {		
		
		currentId = nextIdMin;

		seams[idThread].total_energy += energyImg[currentId].energy;
		seams[idThread].ids[i] = currentId;
		if (colorSeams) {
			imgSrc[currentId].R = 255;
			imgSrc[currentId].B = 0;
			imgSrc[currentId].G = 0;
		}

		int pos = getPosition(currentId, imgProp);
		switch (pos)
		{
		case 0:
		case 2:
			nextIdMin = min(currentId + imgProp->width, currentId + 1 + imgProp->width, energyImg);
			break;
		case 3:
		case 5:
			nextIdMin = min(currentId + imgProp->width, currentId - 1 + imgProp->width, energyImg);
			break;
		case 1:
		case 7:
			break;
		default:
			nextIdMin =  min(min(currentId + imgProp->width, currentId - 1 + imgProp->width, energyImg),
				currentId + 1 + imgProp->width, energyImg);
			break;
		}

		//seams[idThread].total_energy += energyImg[nextIdMin].energy;
		//if(i > 0)
		//	seams[idThread].ids[i] = nextIdMin;

	}

	//if (idThread == 0) {
	//	for (int i = 0; i < imgProp->height; i++) {
	//		printf("%d - ", seams[idThread].ids[i]);
	//	}
	//}
}

void energyMap(energyPixel_t* energyImg, imgProp_t* imgProp) {
	energyMap_ << <imgProp->imageSize / 1024 + 1, 1024 >> > (energyImg, imgProp);
	gpuErrchk(hipDeviceSynchronize());
	//writeBMP_energy("src/assets/images/energy.bmp", energyImg, imgProp);
}



void findSeams(energyPixel_t* energyImg, pixel_t* imgSrc, imgProp_t* imgProp, seam_t *minSeam, seam_t* seams, seam_t* minSeamsPerBlock) {

	//energyPixel_t* img;
	int numBlocks = imgProp->width / 1024 + 1;


	//gpuErrchk(hipMallocManaged(&img, imgProp->imageSize * sizeof(energyPixel_t)));
	//for (int i = 0; i < imgProp->imageSize; i++) {
	//	img[i].pixel.R = energyImg[i].energy;
	//	img[i].pixel.G = energyImg[i].energy;
	//	img[i].pixel.B = energyImg[i].energy;
	//	img[i].energy = energyImg[i].energy;
	//}


	computeSeams << <numBlocks, 1024 >> > (energyImg, imgSrc, seams, imgProp);
	gpuErrchk(hipDeviceSynchronize());

	//pixel_t* img2convert = (pixel_t*)malloc(imgProp->imageSize * sizeof(pixel_t));
	//energy2pixel(img2convert, img, imgProp);
	//writeBMP_pixel(strcat(SOURCE_PATH,"seams_map.bmp"), img2convert, imgProp);

	minArr(numBlocks, 1024, seams, minSeamsPerBlock, imgProp, 1024);
	gpuErrchk(hipDeviceSynchronize());

	*minSeam = minSeamsPerBlock[0];
	for (int i = 1; i < numBlocks; i++) {
		if (minSeamsPerBlock[i].total_energy < minSeam->total_energy) {
			*minSeam = minSeamsPerBlock[i];
		}
	}
	
	//minArr(1, imgProp->width / 1024 + 1, minSeamsPerBlock, minSeam, imgProp, imgProp->width / 1024 + 1);
	//gpuErrchk(hipDeviceSynchronize());


	//dummyMin(seams, *minSeam, imgProp);

	//printf("%d - \n", minSeam[0].total_energy);
	

	//for (int i = 0; i < imgProp->height; i++) {
	//	printf("%d - ", minSeam->ids[i]);
	//}
	//for (int y = 0; y < imgProp->height; y++) {
	//	img[minSeam[0].ids[y]].pixel.R = 0;
	//	img[minSeam[0].ids[y]].pixel.G = 255;
	//	img[minSeam[0].ids[y]].pixel.B = 0;
	//}

	//energy2pixel(img2convert, img, imgProp);
	//writeBMP_pixel(strcat(SOURCE_PATH, "seams_map_minimum.bmp"), img2convert, imgProp);
	//free(img2convert);

	//printf("%d", minSeamPath[0].total_energy);
	//for (int i = 0; i < imgProp->width; i++)
	//gpuErrchk(hipFree(&(seams[i].ids)));

	//for (int i = 0; i < numBlocks; i++)
	//gpuErrchk(hipFree(&minSeamsPerBlock[i].ids));

	//gpuErrchk(hipFree(img));
}

__global__ void removeSeam_(energyPixel_t* energyImg, pixel_t* imgSrc, int* idsToRemove, imgProp_t* imgProp, energyPixel_t* newImageGray, pixel_t* newImageSrc) {
	
	int idThread = blockIdx.x * blockDim.x + threadIdx.x;
	if (idThread < imgProp->imageSize) {
		int idRow = idThread / imgProp->width;
		int idToRemove = idsToRemove[idRow];
		int shift = idThread < idToRemove ? idRow : idRow + 1;

		if (idThread == idToRemove)
			return;

		newImageGray[idThread - shift].energy  = energyImg[idThread].energy;
		newImageGray[idThread - shift].color = energyImg[idThread].color;

		newImageSrc[idThread - shift].R = imgSrc[idThread].R;
		newImageSrc[idThread - shift].G = imgSrc[idThread].G;
		newImageSrc[idThread - shift].B = imgSrc[idThread].B;
	
	}
}

__global__
void updateImageGray_(energyPixel_t* imgGray, energyPixel_t* imgWithoutSeamGray, imgProp_t* imgProp) {
	int idThread = blockIdx.x * blockDim.x + threadIdx.x;
	if (idThread < imgProp->imageSize) {
		imgGray[idThread].energy  = imgWithoutSeamGray[idThread].energy; //si puo commentare?
		imgGray[idThread].color = imgWithoutSeamGray[idThread].color;
	}
}

__global__
void updateImageColored_(pixel_t* imgSrc, pixel_t* imgWithoutSeamSrc, imgProp_t* imgProp) {
	int idThread = blockIdx.x * blockDim.x + threadIdx.x;
	if (idThread < imgProp->imageSize) {
		imgSrc[idThread].R = imgWithoutSeamSrc[idThread].R;
		imgSrc[idThread].G = imgWithoutSeamSrc[idThread].G;
		imgSrc[idThread].B = imgWithoutSeamSrc[idThread].B;
	}
}

void removeSeam(energyPixel_t* imgGray, pixel_t* imgSrc, energyPixel_t* imgWithoutSeamGray, pixel_t* imgWithoutSeamSrc, seam_t* idsToRemove, imgProp_t* imgProp) {

	int newImgSizePixel = imgProp->imageSize - imgProp->height;
	int newFileSize = newImgSizePixel * 3 + 54;
	int numBlocks = newImgSizePixel / 1024 + 1;

	
	removeSeam_ << <numBlocks, 1024 >> > (imgGray, imgSrc, idsToRemove->ids, imgProp, imgWithoutSeamGray, imgWithoutSeamSrc);
	gpuErrchk(hipDeviceSynchronize());

	imgProp->imageSize = newImgSizePixel;
	imgProp->width -= 1;


	updateImageGray_ << <newImgSizePixel/1024 + 1, 1024 >> > (imgGray, imgWithoutSeamGray, imgProp);
	updateImageColored_ << <newImgSizePixel/1024 + 1, 1024 >> > (imgSrc, imgWithoutSeamSrc, imgProp);
	gpuErrchk(hipDeviceSynchronize());
}