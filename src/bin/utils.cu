#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "image_handler.h"
#include "utils.h"
#include "seam_carving.h"

//static const int blockSize = 1024;

__global__ void min_(const seam_t* energiesArray, seam_t* outputArray, imgProp_t* imgProp) {
    int thIdx = threadIdx.x;
    const int myBlockSize = 1024;
    int gthIdx = thIdx + blockIdx.x * myBlockSize;
    __shared__ seam_t shArr[myBlockSize];
    if(gthIdx < imgProp->width)
        shArr[thIdx] = energiesArray[gthIdx];

    int seamsPerBlock = myBlockSize; // 0 < seamsPerBlock < 1024
    
    // si ottiene il numero preciso di seams rimanenti da controllare:
    // per ogni blocco che non sia l'ultimo -> seamsPerBlock = 1024
    // per ultimo blocco -> seamsPerBlock = differenza imgProp->width - (1024 * numBlocchi - 1)
    if (1024 * (blockIdx.x + 1) > imgProp->width)
        seamsPerBlock = imgProp->width - 1024 * blockIdx.x;

    __syncthreads();
    
    int size = seamsPerBlock / 2;
    bool isOdd = seamsPerBlock % 2 == 1;
    if (isOdd) {
        size++;
        if (thIdx < seamsPerBlock / 2)
            shArr[thIdx] = (shArr[thIdx].total_energy < shArr[thIdx + size].total_energy) ? shArr[thIdx] : shArr[thIdx + size];

        size /= 2;
    }
    // get minimum
    for (; size > 0; size /= 2) { //uniform
        if (thIdx < size)
            shArr[thIdx] = (shArr[thIdx].total_energy < shArr[thIdx + size].total_energy) ? shArr[thIdx] : shArr[thIdx + size];
        __syncthreads();
    }

    //save current block's minimum
    if (thIdx == 0) {
        outputArray[blockIdx.x] = shArr[0];
    }
}

void minArr(dim3 gridSize, dim3 blockSize, seam_t* energiesArray, seam_t* outputArray, imgProp_t* imgProp) {
    min_ << <gridSize, blockSize >> > (energiesArray, outputArray, imgProp);
}

void report_gpu_mem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Free = %zu, Total = %zu\n", free, total);
}
