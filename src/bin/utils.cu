#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "image_handler.h"
#include "utils.h"
#include "seam_carving.h"

//static const int blockSize = 1024;

void dummyMin(seam_t* energiesArray, seam_t &output, imgProp_t* imgProp) {
    output = energiesArray[0];
    for (int i = 0; i < imgProp->width; i++) {
        if (output.total_energy > energiesArray[i].total_energy) {
            output = energiesArray[i];
        }
    }

}

__global__ void min_(const seam_t* energiesArray, seam_t* outputArray, imgProp_t* imgProp, int nThreads) {
    int thIdx = threadIdx.x;
    const int myBlockSize = 1024;
    int gthIdx = thIdx + blockIdx.x * myBlockSize;
    extern __shared__ int shArr[];
    int* shared_mins = (int*)shArr;
    int* shared_min_indices = (int*)(&(shArr[nThreads]));

    if (gthIdx < imgProp->width) {
        shared_mins[thIdx] = energiesArray[gthIdx].total_energy;
        shared_min_indices[thIdx] = gthIdx;
    }

    int seamsPerBlock = myBlockSize; // 0 < seamsPerBlock < 1024
    
    // si ottiene il numero preciso di seams rimanenti da controllare:
    // per ogni blocco che non sia l'ultimo -> seamsPerBlock = 1024
    // per ultimo blocco -> seamsPerBlock = differenza imgProp->width - (1024 * numBlocchi - 1)
    if (1024 * (blockIdx.x + 1) > imgProp->width)
        seamsPerBlock = imgProp->width - 1024 * blockIdx.x;

    __syncthreads();
    
    int size = seamsPerBlock / 2;
    bool isOdd = seamsPerBlock % 2 == 1;
    if (isOdd) {
        size++;
        if (thIdx < seamsPerBlock / 2) {
            if (shared_mins[thIdx] > shared_mins[thIdx + size]) {
                shared_mins[thIdx] = shared_mins[thIdx + size];
                shared_min_indices[thIdx] = shared_min_indices[thIdx + size];
            }
        }
        size /= 2;
    }
    // get minimum
    for (; size > 0; size /= 2) { //uniform
        if (thIdx < size) {
            if (shared_mins[thIdx] > shared_mins[thIdx + size]) {
                shared_mins[thIdx] = shared_mins[thIdx + size];
                shared_min_indices[thIdx] = shared_min_indices[thIdx + size];
            }
        }
        __syncthreads();
    }

    //save current block's minimum
    if (thIdx == 0) {
        outputArray[blockIdx.x] = energiesArray[shared_min_indices[0]];
    }
}

void minArr(dim3 gridSize, dim3 blockSize, seam_t* energiesArray, seam_t* outputArray, imgProp_t* imgProp, int nThreads) {
    min_ << <gridSize, blockSize, 1024 * (sizeof(int) + sizeof(int)) >> > (energiesArray, outputArray, imgProp, nThreads);
}

void report_gpu_mem()
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Free = %zu, Total = %zu\n", free, total);
}
