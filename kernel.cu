﻿#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"


struct ImgProp {
	int Hpixels;
	int Vpixels;
	unsigned char HeaderInfo[54];
	unsigned long int Hbytes;
};

struct Pixel {
	unsigned char R;
	unsigned char G;
	unsigned char B;
};

typedef unsigned char pel;    

pel** ReadBMP(char*);  
struct ImgProp ip;

void setupImgProp(ImgProp* ip, FILE* f) {
	pel headerInfo[54];
	fread(headerInfo, sizeof(pel), 54, f);

	int width = *(int*)&headerInfo[18];
	int height = *(int*)&headerInfo[22];
	int rowBytes = (width * 3 + 3) & (~3);

	for (unsigned int i = 0; i < 54; i++)
		ip->HeaderInfo[i] = headerInfo[i];

	ip->Vpixels = height;
	ip->Hpixels = width;
	ip->Hbytes = rowBytes;
}

pel** ReadBMP() {

	//BMP LEGGE I PIXEL NEL FORMATO BGR
	FILE* f = fopen("src/assets/images/castle_bmp.bmp", "rb");
	if (f == NULL) {
		printf("\n\nNOT FOUND\n\n");
		exit(1);
	}

	//extract information from headerInfo
	setupImgProp(&ip, f);
	printf("Input BMP dimension: (%u x %u)\n", ip.Hpixels, ip.Vpixels);

	pel** img;

	hipMallocManaged(&img, ip.Vpixels * sizeof(pel*));
	for (unsigned int i = 0; i < ip.Hpixels; i++)
		hipMallocManaged(&img[i], ip.Hbytes * sizeof(pel));

	for (unsigned int i = 0; i < ip.Vpixels; i++) {
		fread(img[i], sizeof(pel), ip.Hbytes, f);
	}

	fclose(f);
	return img;  // remember to free() it in caller!
}

int main(int argc, char** argv) {

	pel** imgSrc;				

	imgSrc = ReadBMP();

	if (imgSrc == NULL) {
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}

	return 0;
}